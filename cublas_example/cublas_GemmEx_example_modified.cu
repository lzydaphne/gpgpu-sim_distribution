#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 2;
    const int n = 2;
    const int k = 2;
    const int lda = 2;
    const int ldb = 2;
    const int ldc = 2;

    // Use __half for A and B, float for C
    std::vector<__half> A = {__float2half(1.0), __float2half(3.0), __float2half(2.0), __float2half(4.0)};
    std::vector<__half> B = {__float2half(5.0), __float2half(7.0), __float2half(6.0), __float2half(8.0)};
    std::vector<float> C(m * n);
    
    // Use float for alpha and beta to match computation type
    // float alpha = 1.0;
    // float beta = 0.0;
    __half alpha = __float2half(1.0f), beta = __float2half(0.0f);

    __half *d_A = nullptr;
    __half *d_B = nullptr;
    float *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    // Initialize cuBLAS, CUDA
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    // Allocate memory for A, B as __half, and C as float
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(__half) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(__half) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(float) * C.size()));

    // Copy data to device
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(__half) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(__half) * B.size(), hipMemcpyHostToDevice, stream));

    // Perform GEMM operation using hipblasGemmEx
    CUBLAS_CHECK(hipblasGemmEx(
        cublasH, 
        transa, transb, 
        m, n, k, 
        &alpha, 
        d_A, HIP_R_16F, lda, 
        d_B, HIP_R_16F, ldb, 
        &beta, 
        d_C, HIP_R_32F, ldc, 
        HIPBLAS_COMPUTE_32F_PEDANTIC, 
        HIPBLAS_GEMM_DEFAULT
    ));

    // Copy result back to host
    CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, sizeof(float) * C.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Free resources
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
