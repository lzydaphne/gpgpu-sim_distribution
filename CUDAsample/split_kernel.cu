/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <assert.h>
#include <stdio.h>
// #include <curand.h>
// #include <cublas_v2.h>

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


// Define some error checking macros.
#define cudaErrCheck(stat) \
  { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file,
            line);
  }
}

// #define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
// void cublasErrCheck_(cublasStatus_t stat, const char *file, int line) {
//    if (stat != CUBLAS_STATUS_SUCCESS) {
//       f//printf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
//    }
// }

// #define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
// void curandErrCheck_(curandStatus_t stat, const char *file, int line) {
//    if (stat != CURAND_STATUS_SUCCESS) {
//       f//printf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
//    }
// }

#include <hip/hip_fp16.h>
#include <cuda_fp16.hpp>
#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
// #define MATRIX_M 256
// #define MATRIX_N 256
// #define MATRIX_K 256
// #define MATRIX_M 64
// #define MATRIX_N 64
// #define MATRIX_K 64
#define MATRIX_M 128
#define MATRIX_N 128
#define MATRIX_K 128
// Defines to control which parts of the matrix are handled by WMMA or SP
// #define THRESHOLD_ROW 24
// #define THRESHOLD_COL 24
// #define THRESHOLD_ROW 240
// #define THRESHOLD_COL 240
#define THRESHOLD_ROW 112
#define THRESHOLD_COL 112

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// Kernel to calculate the lower right part of the GEMM using SP operations on
// FP16 data
// __global__ void sp_example(float *a, float *b, float *c, int M, int N, int K,
//                            float alpha, float beta) {
//   int row = blockIdx.y * blockDim.y + threadIdx.y;
//   int col = blockIdx.x * blockDim.x + threadIdx.x;

//   if (row >= M || col >= N) return;
//   // Ensure we're within the matrix dimensions
//   if (row < THRESHOLD_ROW || col < THRESHOLD_COL) return;

//   // Adjusted bounds check: only compute certain regions if required
//   //  if (row >= THRESHOLD_ROW && col >= THRESHOLD_COL ) {
//   float sum = 0.0f;
//   //   int range = K / 2  ;
//   // printf("row * K + k = %d\n", row * K + k);
//   // printf("k * N + col = %d\n", k * N + col);
//   for (int k = 0; k < K; ++k) {
//     float a_val = a[row * K + k];  // a is MxK
//     float b_val = b[k * N + col];  // b is KxN
//     sum += alpha * a_val * beta * b_val;

//     // printf("a_val = %f, b_val = %f, sum = %f\n", a_val, b_val, sum);
//   }
//   if (row * N + col < M * N) {  // Check to prevent out-of-bounds access
//     // c[row * N + col] = alpha * sum + beta * c[row * N + col];
//     c[row * N + col] += sum;
//   }
// }

// // Kernel to calculate the lower right part of the GEMM using SP operations on
// // FP16 data
__global__ void sp_example(half *a, half *b, float *c, int M, int N, int K,
                           float alpha, float beta) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row >= M || col >= N) return;
  // Ensure we're within the matrix dimensions
  if (row < THRESHOLD_ROW || col < THRESHOLD_COL) return;

  // Adjusted bounds check: only compute certain regions if required
  //  if (row >= THRESHOLD_ROW && col >= THRESHOLD_COL ) {
  // half sum = 0.0f;
  half sum = 0.0f;
  //   int range = K / 2  ;
  // printf("row * K + k = %d\n", row * K + k);
  // printf("k * N + col = %d\n", k * N + col);
  for (int k = 0; k < K; ++k) {
    half a_val = a[row * K + k];  // a is MxK
    half b_val = b[k * N + col];  // b is KxN
    // sum += alpha * a_val + beta * b_val;
    // sum += alpha * __half2float(a_val) + beta * __half2float(b_val);

    //* caculation down below perform slower
    sum = __hadd(sum, __hadd(__hmul(alpha, a_val), __hmul(beta, b_val)));

    // printf("a_val = %f, b_val = %f, sum = %f\n", a_val, b_val, sum);
  }
  if (row * N + col < M * N) {  // Check to prevent out-of-bounds access
    // c[row * N + col] = alpha * sum + beta * c[row * N + col];
    c[row * N + col] += __half2float(sum);
  }
  // __syncwarp();
}

// CD kernel
// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration
// purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K,
                             float alpha, float beta) {
  // Leading dimensions. Packed with no transpositions.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
  // printf("wmma_example: warpM = %d, warpN = %d\n", warpM, warpN);

  //! Only skip processing if both row and column indices exceed their thresholds
  if (warpM * WMMA_M >= THRESHOLD_ROW && warpN * WMMA_N >= THRESHOLD_COL)
    return;

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  // printf("----start--forloop-- threadIdx.x: %d, threadIdx.y: %d\n",
  // threadIdx.x, threadIdx.y);
  for (int i = 0; i < K; i += WMMA_K) {
    // printf("----start--target---sp--\n");

    int aRow = warpM * WMMA_M;
    int aCol = i;

    int bRow = i;
    int bCol = warpN * WMMA_N;

  // Ensure only the first thread in each warp prints
    // if (threadIdx.x % warpSize == 0 ) {
    //   printf("aRow: %d, aCol: %d, bRow: %d, bCol: %d, i: %d\n", aRow, aCol,
    //          bRow, bCol, i);
    // }
    // printf("aRow: %d, aCol: %d, bRow: %d, bCol: %d, i: %d\n", aRow, aCol,
    // bRow, bCol, i); printf("---end---target---sp--\n");

    // Bounds checking
    if (aRow < M && aCol < K && bRow < K && bCol < N) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);

      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cRow = warpM * WMMA_M;
  int cCol = warpN * WMMA_N;
  // Ensure only the first thread in each warp prints

  if (cRow < M && cCol < N) {
    // if (threadIdx.x % warpSize == 0 ) {
        // printf("cRow= %d, cCol = %d\n", cRow, cCol);
    // }
    wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc,
                           wmma::mem_col_major);

#pragma unroll
    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc,
                            wmma::mem_col_major);

    // Ensure only the first thread in each warp prints
    // if (threadIdx.x % warpSize == 0 ) {
    //     printf("wmma_example: warpM = %d, warpN = %d\n", warpM, warpN);
    // }
  }
}

__global__ void convertFp32ToFp16(half *out, float *in, int n) {
  // count cycles

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n) {
    out[idx] = in[idx];
  }
}

int main(int argc, char *argv[]) {
  // Initialize the problem
  int nkernels = 2;             // number of concurrent kernels
  int nstreams = nkernels + 1;  // use one more stream than concurrent kernel
  // int nbytes = nkernels * sizeof(float_t);  // number of data bytes
  float kernel_time = 10;             // time the kernel should run in ms
  float elapsed_time, elapsed_time2;  // timing variables
  int cuda_device = 0;

  hipDeviceProp_t deviceProp;
  cudaErrCheck(hipGetDevice(&cuda_device));

  cudaErrCheck(hipGetDeviceProperties(&deviceProp, cuda_device));

  if ((deviceProp.concurrentKernels == 0)) {
    printf("> GPU does not support concurrent kernel execution\n");
    printf("  CUDA kernel runs will be serialized\n");
  } else {
    printf("concurrent kernel: %d\n", deviceProp.concurrentKernels);
  }

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
  //--------------------
  // cuda core
  int N = MATRIX_N;  // Define the size of the matrix
  size_t size = N * N * sizeof(float_t);
  float *h_A, *h_B, *h_C;           // host copies of A, B, C
  float *d_A, *d_B, *d_C;           // device copies of A, B, C

  // float *a_fp32;
  // float *b_fp32;
  half *a_fp16;
  half *b_fp16;
  // //printf("WMMA Example2\n");

  float *c;
  //    float *c_cublas;
  float *c_wmma;

  //    float *c_host_cublas;
  float *c_host_wmma;
  // //printf("WMMA Example3\n");

  // cuda core: Allocate space for host copies and setup values
  cudaErrCheck(hipHostMalloc((void **)&h_A, size, hipHostMallocDefault));
  cudaErrCheck(hipHostMalloc((void **)&h_B, size, hipHostMallocDefault));
  cudaErrCheck(hipHostMalloc((void **)&h_C, size, hipHostMallocDefault));
  //    h_A = (float *)malloc(size);
  //    h_B = (float *)malloc(size);
  //    h_C = (float *)malloc(size);

  // Allocate space for device copies
  cudaErrCheck(hipMalloc((void **)&d_A, size));
  cudaErrCheck(hipMalloc((void **)&d_B, size));
  cudaErrCheck(hipMalloc((void **)&d_C, size));
  // cudaErrCheck(cudaMalloc((void **)&d_tmpC, size));
  //    cudaMalloc((void **)&d_A, size);
  //    cudaMalloc((void **)&d_B, size);
  //    cudaMalloc((void **)&d_C, size);
  cudaErrCheck(
      hipMalloc((void **)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
  cudaErrCheck(
      hipMalloc((void **)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

  // Initialize matrices A and B with random values
  for (int i = 0; i < N * N; i++) {
    h_A[i] = (float)rand() / (float)RAND_MAX *
             100.0;  // Assign a random float value between 0 and 100
    h_B[i] = (float)rand() / (float)RAND_MAX *
             100.0;  // Assign a random float value between 0 and 100
    h_C[i] = 1.5;
  }
  //    initializeMatrix(h_A, N);
  //    initializeMatrix(h_B, N);

  // stream create
  // allocate and initialize an array of stream handles
  hipStream_t *streams =
      (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  hipStreamCreateWithPriority(&streams[0], hipStreamNonBlocking, 0);
  hipStreamCreateWithPriority(&streams[1], hipStreamNonBlocking, 1);
  hipStreamCreateWithPriority(&streams[2], hipStreamNonBlocking, 2);
  // for (int i = 1; i < nstreams; i++) {
  //   cudaErrCheck(cudaStreamCreate(&(streams[i])));
  // }

  // Copy inputs to device
  cudaErrCheck(
      hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, streams[0]));
  cudaErrCheck(
      hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, streams[1]));
  cudaErrCheck(
      hipMemcpyAsync(d_C, h_C, size, hipMemcpyHostToDevice, streams[2]));
  // cudaErrCheck(
  //     cudaMemcpyAsync(d_tmpC, h_C, size, cudaMemcpyHostToDevice, streams[2]));
  // cudaErrCheck(cudaMemcpy(d_A, h_A, size, cudaMemcpyHostToDevice));
  // cudaErrCheck(cudaMemcpy(d_B, h_B, size, cudaMemcpyHostToDevice));
  // cudaErrCheck(cudaMemcpy(d_C, h_C, size, cudaMemcpyHostToDevice));
  // cudaErrCheck(cudaMemcpy(d_tmpC, h_C, size, cudaMemcpyHostToDevice));

//   cudaEvent_t startWMMA, startMatrix;
//   cudaEvent_t stopWMMA, stopMatrix;

//   cudaErrCheck(cudaEventCreate(&startWMMA));
//   cudaErrCheck(cudaEventCreate(&stopWMMA));
  // cudaErrCheck(cudaEventCreate(&startMatrix));
  // cudaErrCheck(cudaEventCreate(&stopMatrix));

  //    cudaErrCheck(cudaMalloc((void **)&a_fp32, MATRIX_M * MATRIX_K *
  //    sizeof(float))); cudaErrCheck(cudaMalloc((void **)&b_fp32, MATRIX_K *
  //    MATRIX_N * sizeof(float)));

  //    cudaErrCheck(cudaMalloc((void **)&c, MATRIX_M * MATRIX_N *
  //    sizeof(float))); cudaErrCheck(cudaMalloc((void**)&c_cublas, MATRIX_M *
  //    MATRIX_N * sizeof(float)));
  //    cudaErrCheck(cudaMalloc((void **)&c_wmma, MATRIX_M * MATRIX_N *
  //    sizeof(float)));

  //    c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
  //    c_host_wmma = (float *)malloc(MATRIX_M * MATRIX_N * sizeof(float));

  //    curandErrCheck(curandCreateGenerator(&gen, CURAND_RNG_PSEUDO_DEFAULT));
  //    curandErrCheck(curandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

  //    curandErrCheck(curandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
  //    curandErrCheck(curandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

  // curand doesn't currently support fp16 so we generate in fp32 and convert to
  // fp16.
  //! concurrent

  printf("Converting to fp16...a_fp16\n");
  // printf("Current cycle time:");
  // convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 31) / 32, 32>>>(a_fp16, d_A,
  // MATRIX_M * MATRIX_K);
//   convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 31) / 32, 32, 0, 0>>>(
//       a_fp16, d_A, MATRIX_M * MATRIX_K);
  // convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 31) / 32, 32, 0, streams[0]>>>(
  convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 31) / 32, 32,0,0>>>(
      a_fp16, d_A, MATRIX_M * MATRIX_K);
  // convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 63) / 64, 64, 0,
  // streams[0]>>>(a_fp16, h_A, MATRIX_M * MATRIX_K);
  // convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 127) / 128, 128, 0,
  // streams[0]>>>(a_fp16, h_A, MATRIX_M * MATRIX_K);
  // convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 255) / 256, 256,0,
  // streams[0]>>>(a_fp16, a_fp32, MATRIX_M * MATRIX_K);
  //* sequential
  //    convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 127) / 128, 128, 0,
  //    0>>>(a_fp16, h_A, MATRIX_M * MATRIX_K);

  // convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 255) / 256, 256>>>(a_fp16,
  // a_fp32, MATRIX_M * MATRIX_K);
  // printf("Current cycle time done:");
  // //printf("Current cycle time: %f\n", getCycleTime());
  //! concurrent
  printf("Converting to fp16...b_fp16\n");
  // printf("Current cycle time:");
//   convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 31) / 32, 32>>>(b_fp16, d_B,
//   MATRIX_K * MATRIX_N);
//   convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 31) / 32, 32, 0, 0>>>(
//       b_fp16, d_B, MATRIX_K * MATRIX_N);
  convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 31) / 32, 32,0,0>>>(
      b_fp16, d_B, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 31) / 32, 32, 0, streams[1]>>>(
  //     b_fp16, d_B, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 63) / 64, 64, 0,
  // streams[1]>>>(b_fp16, h_B, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 127) / 128, 128, 0,
  // streams[1]>>>(b_fp16, h_B, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 255) / 256, 256,0,
  // streams[1]>>>(b_fp16, h_B, MATRIX_K * MATRIX_N);
  //* sequential
  //    convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 127) / 128, 128, 0,
  //    0>>>(b_fp16, h_B, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 255) / 256, 256>>>(b_fp16,
  // b_fp32, MATRIX_K * MATRIX_N);
  // printf("Converting to fp16... DONE\n");

  //    curandErrCheck(curandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));

  //    curandErrCheck(curandDestroyGenerator(gen));

  //    cudaErrCheck(cudaMemcpy(c_cublas, c, MATRIX_M * MATRIX_N *
  //    sizeof(float), cudaMemcpyDeviceToDevice));
  //    cudaErrCheck(cudaMemcpy(c_wmma, h_C, MATRIX_M * MATRIX_N *
  //    sizeof(float), cudaMemcpyHostToDevice));

  float alpha = 2.0f;
  float beta = 2.0f;
  half alpha_fp16 = 2.0;
  half beta_fp16 = 2.0;

  // printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M,
  // MATRIX_N, MATRIX_K, alpha, beta);

  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
  // blockDim.x = 32;
  // blockDim.y = 1;
  blockDim.x = 32;
  blockDim.y = 8;
  // blockDim.x = 64;
  // blockDim.y = 8;
  //    blockDim.x = 128;
  //    blockDim.y = 1;
  // blockDim.x = 64;
  // blockDim.y = 1;
  // blockDim.x = 128;
  // blockDim.y = 4;

  // gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 256 - 1)) / (WMMA_M *
  // blockDim.x / 256); gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) /
  // (WMMA_N * blockDim.y); gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 128 -
  // 1)) / (WMMA_M * blockDim.x / 128); gridDim.y = (MATRIX_N + WMMA_N *
  // blockDim.y - 1) / (WMMA_N * blockDim.y);
  gridDim.x =
      (MATRIX_M + (WMMA_M * blockDim.x / 64 - 1)) / (WMMA_M * blockDim.x / 64);
  gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
  // gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M *
  // blockDim.x / 32); gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) /
  // (WMMA_N * blockDim.y);
  printf("wmma_example: gridDim.x = %d, gridDim.y = %d\n", gridDim.x,
         gridDim.y);

//   cudaErrCheck(cudaEventRecord(startWMMA));
  // wmma_example<<<gridDim, blockDim>>>(a_fp16, b_fp16, c_wmma, MATRIX_M,
  // MATRIX_N, MATRIX_K, alpha, beta); matrixMulKernel<<<gridDim,
  // blockDim>>>(d_A, d_B, d_C, N);

  //* sequential
  //    wmma_example<<<gridDim, blockDim, 0, 0>>>(a_fp16, b_fp16, d_C, MATRIX_M,
  //    MATRIX_N, MATRIX_K, alpha, beta); matrixMulKernel<<<gridDim, blockDim,
  //    0, 0>>>(d_A, d_B, d_C, N);
  //! concurrent
  // cudaDeviceSynchronize();
  // cudaStreamSynchronize(streams[0]);
  // wmma_example<<<gridDim, blockDim, 0, 0>>>(
  //     a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  wmma_example<<<gridDim, blockDim, 0, streams[0]>>>(
      a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

  dim3 sp_blockDim(16,16);  // Commonly used block size for matrix multiplication
  dim3 sp_gridDim;
  sp_gridDim.x = (N + sp_blockDim.x - 1) / sp_blockDim.x;
  sp_gridDim.y = (N + sp_blockDim.y - 1) / sp_blockDim.y;
  printf("sp_example's : gridDim.x = %d, gridDim.y = %d\n", sp_gridDim.x,
         sp_gridDim.y);
  // sp_example<<<sp_gridDim, sp_blockDim, 0,  streams[0]>>>(
  //     a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
    sp_example<<<sp_gridDim, sp_blockDim, 0, streams[1]>>>(
        a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha_fp16, beta_fp16);
  // wmma_example<<<gridDim, blockDim, 0, streams[1]>>>(
  //     a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

  // cudaDeviceSynchronize();
  //   cudaErrCheck(cudaEventRecord(stopWMMA));
  //   printf("Running with cudaEventSynchronize...\n");
  //   cudaErrCheck(cudaEventSynchronize(stopWMMA));
  //   printf("Running with cudaEventSynchronize-end...\n");

  //   cudaErrCheck(cudaEventElapsedTime(&elapsed_time, startWMMA, stopWMMA));
  //   printf("Measured time for sample = %.3f ms\n", elapsed_time);
  //    printf("Running with wmma...done\n");


  // cudaErrCheck(cudaDeviceSynchronize());

  // cudaErrCheck(cudaEventRecord(startMatrix,streams[1]));
  // printf("Running with matrixMulKernel\n");

  // dim3 blockDim_matrix (16, 16);  // Commonly used block size for matrix
  // multiplication dim3 gridDim_matrix; gridDim_matrix.x = (N +
  // blockDim_matrix.x - 1) / blockDim_matrix.x; gridDim_matrix.y = (N +
  // blockDim_matrix.y - 1) / blockDim_matrix.y;

  // printf("matrixMulKernel's : gridDim.x = %d, gridDim.y = %d\n",
  // gridDim_matrix.x, gridDim_matrix.y); matrixMulKernel<<<gridDim_matrix,
  // blockDim_matrix, 0, streams[1]>>>(d_A, d_B, d_C, N);

  // cudaErrCheck(cudaEventRecord(stopMatrix,streams[1]));
  // cudaErrCheck(cudaEventSynchronize(stopMatrix));
  // cudaErrCheck(cudaEventElapsedTime(&elapsed_time2, startMatrix,
  // stopMatrix)); printf("Measured time for sample-2 = %.3f ms\n",
  // elapsed_time2 );
  //    printf("Running with matrixMulKernel...done\n");

  hipStreamSynchronize(streams[0]);
  hipStreamSynchronize(streams[1]);
  hipStreamSynchronize(streams[2]);

  hipDeviceSynchronize();

  // Error checking
  printf("\nChecking results...\n");
  //    cudaErrCheck(cudaMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N *
  //    sizeof(float), cudaMemcpyDeviceToHost));

  //* sequential
  //    cudaErrCheck(cudaMemcpy(h_C, d_C, size, cudaMemcpyDeviceToHost));
  //! concurrent
  cudaErrCheck(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
  // cudaErrCheck(cudaMemcpyAsync(h_C, d_C, size,
  // cudaMemcpyDeviceToHost,streams[nstreams - 1]));
  //    cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N *
  //    sizeof(float), cudaMemcpyDeviceToHost));

  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  hipStreamDestroy(streams[2]);
  free(streams);

//   cudaErrCheck(cudaEventDestroy(startWMMA));
//   cudaErrCheck(cudaEventDestroy(stopWMMA));
  // cudaErrCheck(cudaEventDestroy(startMatrix));
  // cudaErrCheck(cudaEventDestroy(stopMatrix));

  //   cudaErrCheck(cudaEventDestroy(startcublas));
  //   cudaErrCheck(cudaEventDestroy(stopcublas));

  //    cudaErrCheck(cudaFree(a_fp32));
  //    cudaErrCheck(cudaFree(b_fp32));
  cudaErrCheck(hipFree(a_fp16));
  cudaErrCheck(hipFree(b_fp16));

  //    cudaErrCheck(cudaFree(c));
  //    cudaErrCheck(cudaFree(c_cublas));
  //    cudaErrCheck(cudaFree(c_wmma));

  // cuda core
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  // cudaFree(d_tmpC);
  //* Pinned Memory
  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(h_C);

  //    free(c_host_cublas);
  //    free(c_host_wmma);

  //    cudaErrCheck(cudaDeviceReset());
  //    return 0;
  exit(EXIT_SUCCESS);
}