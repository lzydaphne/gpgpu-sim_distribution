/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// #include <cublas_v2.h>
// #include <curand.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

// Define some error checking macros.
#define cudaErrCheck(stat) \
  { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file,
            line);
    printf("CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
  }
}

// #define cublasErrCheck(stat) \
//   { cublasErrCheck_((stat), __FILE__, __LINE__); }
// void cublasErrCheck_(cublasStatus_t stat, const char *file, int line) {
//   if (stat != CUBLAS_STATUS_SUCCESS) {
//     fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
//   }
// }

// #define curandErrCheck(stat) \
//   { curandErrCheck_((stat), __FILE__, __LINE__); }
// void curandErrCheck_(curandStatus_t stat, const char *file, int line) {
//   if (stat != CURAND_STATUS_SUCCESS) {
//     fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
//   }
// }

#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M 64
#define MATRIX_N 64
#define MATRIX_K 64

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration
// purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K,
                             float alpha, float beta) {
  // Leading dimensions. Packed with no transpositions.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  for (int i = 0; i < K; i += WMMA_K) {
    int aRow = warpM * WMMA_M;
    int aCol = i;

    int bRow = i;
    int bCol = warpN * WMMA_N;

    // Bounds checking
    if (aRow < M && aCol < K && bRow < K && bCol < N) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cRow = warpM * WMMA_M;
  int cCol = warpN * WMMA_N;

  if (cRow < M && cCol < N) {
    wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc,
                           wmma::mem_col_major);

#pragma unroll
    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc,
                            wmma::mem_col_major);
  }
}

// __global__ void convertFp32ToFp16(half *out, float *in, int n) {
//   int idx = blockDim.x * blockIdx.x + threadIdx.x;
//   if (idx < n) {
//     out[idx] = in[idx];
//   }
// }
__global__ void convertFp32ToFp16(half *out, float *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __float2half_rn(in[idx]);
    }
}
int main(int argc, char *argv[]) {
  float *a_fp32;
  float *b_fp32;
  half *a_fp16;
  half *b_fp16;

  float *c_wmma;
  float *c_host_wmma;

//! Host memory allocation
float *h_a_fp32 = (float *)malloc(MATRIX_M * MATRIX_K * sizeof(float));
float *h_b_fp32 = (float *)malloc(MATRIX_K * MATRIX_N * sizeof(float));
float *h_c = (float *)malloc(MATRIX_M * MATRIX_N * sizeof(float));
//! Host memory for the result
c_host_wmma = (float *)malloc(MATRIX_M * MATRIX_N * sizeof(float));
if (!c_host_wmma) {
      fprintf(stderr, "Host memory allocation for result failed\n");
      exit(1);
  }

// Check for successful host memory allocation
  if (!h_a_fp32 || !h_b_fp32 || !h_c || !c_host_wmma) {
      printf("Host memory allocation failed\n");
      return -1;
  }
//! Fill host memory with values
  for (int i = 0; i < MATRIX_M * MATRIX_K; ++i) {
      h_a_fp32[i] = float(i % 255 - 127) / 127;
  } 
  for (int i = 0; i < MATRIX_K * MATRIX_N ; ++i) {
      h_b_fp32[i] = float(i % 255 - 127) / 127;
  }
  for (int i = 0; i < MATRIX_M * MATRIX_N ; ++i) {
      h_c[i] = float(i % 255 - 127) / 127;
  }

//! Device memory allocation
  cudaErrCheck(
      hipMalloc((void **)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
  cudaErrCheck(
      hipMalloc((void **)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
  cudaErrCheck(
      hipMalloc((void **)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
  cudaErrCheck(
      hipMalloc((void **)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void **)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));


 

//! Copy host data to device
cudaErrCheck(hipMemcpy(a_fp32, h_a_fp32, MATRIX_M * MATRIX_K * sizeof(float), hipMemcpyHostToDevice));
cudaErrCheck(hipMemcpy(b_fp32, h_b_fp32, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
cudaErrCheck(hipMemcpy(c_wmma, h_c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));


//   curandErrCheck(curandCreateGenerator(&gen, CURAND_RNG_PSEUDO_DEFAULT));
//   curandErrCheck(curandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

//   curandErrCheck(curandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
//   curandErrCheck(curandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));
//! Fill in the numbers
//* cannot directly access device memory from host code
// for (int i = 0; i < MATRIX_M * MATRIX_K; ++i) {
//       a_fp16[i] = __float2half_rn(a_fp32[i]);
// } 
// for (int i = 0; i < MATRIX_K * MATRIX_N ; ++i) {
//     b_fp16[i] = __float2half_rn(b_fp32[i]);
// }

convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 255) / 256, 256>>>(
    a_fp16, a_fp32, MATRIX_M * MATRIX_K);
convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 255) / 256, 256>>>(
    b_fp16, b_fp32, MATRIX_K * MATRIX_N);

//! For c matrix

  float alpha = 2.0f;
  float beta = 2.0f;

  printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M,
         MATRIX_N, MATRIX_K, alpha, beta);

  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
  blockDim.x = 128;
  blockDim.y = 4;

  gridDim.x =
      (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
  gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  printf("Running with wmma...\n");
  
  wmma_example<<<gridDim, blockDim>>>(a_fp16, b_fp16, c_wmma, MATRIX_M,
                                      MATRIX_N, MATRIX_K, alpha, beta);

 
  // Error checking
hipError_t err = hipDeviceSynchronize();
if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
}

  printf("\nChecking results...\n");
  cudaErrCheck(hipMemcpy(c_host_wmma, c_wmma,
                          MATRIX_M * MATRIX_N * sizeof(float),
                          hipMemcpyDeviceToHost));

   // Free host memory
    free(h_a_fp32);
    free(h_b_fp32);
    free(h_c);
    free(c_host_wmma);

    // Free device memory
    cudaErrCheck(hipFree(a_fp32));
    cudaErrCheck(hipFree(b_fp32));
    cudaErrCheck(hipFree(a_fp16));
    cudaErrCheck(hipFree(b_fp16));
    cudaErrCheck(hipFree(c_wmma));

    cudaErrCheck(hipDeviceReset());
  return 0;
}
