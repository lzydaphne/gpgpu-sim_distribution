/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


// Define some error checking macros.
#define cudaErrCheck(stat) \
  { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file,
            line);
  }
}


#include <hip/hip_fp16.h>
#include <cuda_fp16.hpp>
#include <mma.h>
using namespace nvcuda;

#define MATRIX_M 20480
#define MATRIX_N 20480
#define MATRIX_K 20480
#define THRESHOLD_ROW 20464
#define THRESHOLD_COL 20464
// #define MATRIX_M 10240
// #define MATRIX_N 10240
// #define MATRIX_K 10240
// #define THRESHOLD_ROW 10224
// #define THRESHOLD_COL 10224
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__global__ void sp_example(half *a, half *b, float *c, int M, int N, int K,
                           float alpha, float beta) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < THRESHOLD_ROW || col < THRESHOLD_COL) return;
  if (row >= M || col >= N) return;
 
  half sum = 0.0f;
  for (int k = 0; k < K; ++k) {
    half a_val = a[row * K + k];  // a is MxK
    half b_val = b[k * N + col];  // b is KxN

    //* caculation down below perform slower
    sum = __hadd(sum, __hadd(__hmul(alpha, a_val), __hmul(beta, b_val)));

  }
c[row * N + col] += __half2float(sum);
}

// CD kernel
// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration
// purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K,
                             float alpha, float beta) {

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
  //! Only skip processing if both row and column indices exceed their thresholds
  if(warpM * WMMA_M >= THRESHOLD_ROW && warpN * WMMA_N >= THRESHOLD_COL) return;

  // Leading dimensions. Packed with no transpositions.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  for (int i = 0; i < K; i += WMMA_K) {
    // printf("----start--target---sp--\n");
    int aRow = warpM * WMMA_M;
    int aCol = i;

    int bRow = i;
    int bCol = warpN * WMMA_N;

    // Bounds checking
    if (aRow < M && aCol < K && bRow < K && bCol < N) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);

      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cRow = warpM * WMMA_M;
  int cCol = warpN * WMMA_N;
  // Ensure only the first thread in each warp prints

  if (cRow < M && cCol < N) {
   wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

#pragma unroll
    // printf("wmma_example: cRow = %d, cCol = %d\n", cRow, cCol);
    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];

    /// Store the output
    wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);

    } 
  }
}

__global__ void convertFp32ToFp16(half *out, float *in, int n) {
  // count cycles

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n) {
    out[idx] = in[idx];
  }
}

int main(int argc, char *argv[]) {
  // Initialize the problem
  int nkernels = 2;             // number of concurrent kernels
  int nstreams = nkernels + 1;  // use one more stream than concurrent kernel
  // int nbytes = nkernels * sizeof(float_t);  // number of data bytes
  float kernel_time = 10;             // time the kernel should run in ms
  float elapsed_time, elapsed_time2;  // timing variables
  int cuda_device = 0;

  hipDeviceProp_t deviceProp;
  cudaErrCheck(hipGetDevice(&cuda_device));

  cudaErrCheck(hipGetDeviceProperties(&deviceProp, cuda_device));

  if ((deviceProp.concurrentKernels == 0)) {
    printf("> GPU does not support concurrent kernel execution\n");
    printf("  CUDA kernel runs will be serialized\n");
  } else {
    printf("concurrent kernel: %d\n", deviceProp.concurrentKernels);
  }

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    //evaluate the performance
    hipEvent_t start, stop;
    float elapsedTime;


  // cuda core
  int N = MATRIX_N;  // Define the size of the matrix
  size_t size = N * N * sizeof(float_t);
  float *h_A, *h_B, *h_C;           // host copies of A, B, C
  float *d_A, *d_B, *d_C;           // device copies of A, B, C

  // float *a_fp32;
  // float *b_fp32;
  half *a_fp16;
  half *b_fp16;
  // //printf("WMMA Example2\n");

  float *c;
  //    float *c_cublas;
  float *c_wmma;

  //    float *c_host_cublas;
  float *c_host_wmma;
  // //printf("WMMA Example3\n");

  // cuda core: Allocate space for host copies and setup values
  cudaErrCheck(hipHostMalloc((void **)&h_A, size, hipHostMallocDefault));
  cudaErrCheck(hipHostMalloc((void **)&h_B, size, hipHostMallocDefault));
  cudaErrCheck(hipHostMalloc((void **)&h_C, size, hipHostMallocDefault));

  // Allocate space for device copies
  cudaErrCheck(hipMalloc((void **)&d_A, size));
  cudaErrCheck(hipMalloc((void **)&d_B, size));
  cudaErrCheck(hipMalloc((void **)&d_C, size));
  cudaErrCheck(
      hipMalloc((void **)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
  cudaErrCheck(
      hipMalloc((void **)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

  // Initialize matrices A and B with random values
  for (int i = 0; i < N * N; i++) {
    h_A[i] = (float)rand() / (float)RAND_MAX *
             10.0;  // Assign a random float value between 0 and 100
    h_B[i] = (float)rand() / (float)RAND_MAX *
             10.0;  // Assign a random float value between 0 and 100
    h_C[i] = 1.5;
  }
  // stream create
  // allocate and initialize an array of stream handles
  hipStream_t *streams =
      (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  hipStreamCreateWithPriority(&streams[0], hipStreamNonBlocking, 0);
  hipStreamCreateWithPriority(&streams[1], hipStreamNonBlocking, 1);
  hipStreamCreateWithPriority(&streams[2], hipStreamNonBlocking, 2);
  // for (int i = 1; i < nstreams; i++) {
  //   cudaErrCheck(cudaStreamCreate(&(streams[i])));
  // }

  // Copy inputs to device
  cudaErrCheck(
      hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, streams[0]));
  cudaErrCheck(
      hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, streams[1]));
  cudaErrCheck(
      hipMemcpyAsync(d_C, h_C, size, hipMemcpyHostToDevice, streams[2]));
  
  //! concurrent

  printf("Converting to fp16...a_fp16\n");
  convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 31) / 32, 32,0,0>>>(
      a_fp16, d_A, MATRIX_M * MATRIX_K);
  //! concurrent
  printf("Converting to fp16...b_fp16\n");
 
  convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 31) / 32, 32,0,0>>>(
      b_fp16, d_B, MATRIX_K * MATRIX_N);
 

  float alpha = 2.0f;
  float beta = 2.0f;
  half alpha_fp16 = 2.0;
  half beta_fp16 = 2.0;

  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  blockDim.x = 128;
  blockDim.y = 4;
 
  gridDim.x =
      (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
  gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);


//* Create events for timing
hipEventCreate(&start);
hipEventCreate(&stop);
 hipEventRecord(start, 0);


//! wmma kernel
// wmma_example<<<gridDim, blockDim, 0, 0>>>(
//       a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
wmma_example<<<gridDim, blockDim, 0, streams[0]>>>(
      a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);


  dim3 sp_blockDim(16,16);  // Commonly used block size for matrix multiplication
  dim3 sp_gridDim;
  sp_gridDim.x = (N + sp_blockDim.x - 1) / sp_blockDim.x;
  sp_gridDim.y = (N + sp_blockDim.y - 1) / sp_blockDim.y;
  printf("sp_example's : gridDim.x = %d, gridDim.y = %d\n", sp_gridDim.x,
         sp_gridDim.y);

//! sp kernel
sp_example<<<sp_gridDim, sp_blockDim, 0, streams[1]>>>(
        a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha_fp16, beta_fp16);
    
  

  hipStreamSynchronize(streams[0]);
  hipStreamSynchronize(streams[1]);


hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime, start, stop);
printf("Time for WMMA kernel: %f ms\n", elapsedTime);
hipEventDestroy(start);
hipEventDestroy(stop);

 //* use to prevent segfault!!!
  hipDeviceSynchronize();

  // Error checking
  printf("\nChecking results...\n");

  cudaErrCheck(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));


  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  hipStreamDestroy(streams[2]);
  free(streams);

  cudaErrCheck(hipFree(a_fp16));
  cudaErrCheck(hipFree(b_fp16));


  // cuda core
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  // cudaFree(d_tmpC);
  //* Pinned Memory
  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(h_C);

  exit(EXIT_SUCCESS);
}