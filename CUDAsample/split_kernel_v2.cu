/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <assert.h>
#include <stdio.h>
// #include <curand.h>
// #include <cublas_v2.h>

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


// Define some error checking macros.
#define cudaErrCheck(stat) \
  { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file,
            line);
  }
}


#include <hip/hip_fp16.h>
#include <cuda_fp16.hpp>
#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
// #define MATRIX_M 32
// #define MATRIX_N 32
// #define MATRIX_K 32
#define MATRIX_M 32
#define MATRIX_N 32
#define MATRIX_K 32
// #define MATRIX_M 7680
// #define MATRIX_N 7680
// #define MATRIX_K 7680
// Defines to control which parts of the matrix are handled by WMMA or SP
// #define THRESHOLD_ROW 24
// #define THRESHOLD_COL 24
#define THRESHOLD_ROW 16
#define THRESHOLD_COL 16
// #define THRESHOLD_ROW 240
// #define THRESHOLD_COL 240
// #define THRESHOLD_ROW 112
// #define THRESHOLD_COL 112
// #define THRESHOLD_ROW 7664
// #define THRESHOLD_COL 7664
// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__global__ void sp_example(half *a, half *b, float *c, int M, int N, int K,
                           float alpha, float beta) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < THRESHOLD_ROW || col < THRESHOLD_COL) return;
  if (row >= M || col >= N) return;
  // Ensure we're within the matrix dimensions

  // Adjusted bounds check: only compute certain regions if required
  //  if (row >= THRESHOLD_ROW && col >= THRESHOLD_COL ) {
  // half sum = 0.0f;
  half sum = 0.0f;
  //   int range = K / 2  ;
  // printf("row * K + k = %d\n", row * K + k);
  // printf("k * N + col = %d\n", k * N + col);
  for (int k = 0; k < K; ++k) {
    half a_val = a[row * K + k];  // a is MxK
    half b_val = b[k * N + col];  // b is KxN
    // sum += alpha * a_val + beta * b_val;
    // sum += alpha * __half2float(a_val) + beta * __half2float(b_val);

    //* caculation down below perform slower
    sum = __hadd(sum, __hadd(__hmul(alpha, a_val), __hmul(beta, b_val)));

    // printf("a_val = %f, b_val = %f, sum = %f\n", a_val, b_val, sum);
  }
c[row * N + col] += __half2float(sum);
// if (threadIdx.x % warpSize == 0 ) {
//   printf("row = %d, col = %d, sum = %f\n", row, col, c[row * N + col]);
// }
//   if (row * N + col < M * N) {  // Check to prevent out-of-bounds access
//     // c[row * N + col] = alpha * sum + beta * c[row * N + col];
//   }
  // __syncwarp();
}

// CD kernel
// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration
// purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K,
                             float alpha, float beta) {

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
// if (threadIdx.x % warpSize == 0 ) {
//     printf("wmma_example: warpM = %d, warpN = %d\n", warpM, warpN);
// }
  //! Only skip processing if both row and column indices exceed their thresholds
  if(warpM * WMMA_M >= THRESHOLD_ROW && warpN * WMMA_N >= THRESHOLD_COL) return;

  // Leading dimensions. Packed with no transpositions.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  // printf("----start--forloop-- threadIdx.x: %d, threadIdx.y: %d\n",
  // threadIdx.x, threadIdx.y);
  for (int i = 0; i < K; i += WMMA_K) {
    // printf("----start--target---sp--\n");

    int aRow = warpM * WMMA_M;
    int aCol = i;

    int bRow = i;
    int bCol = warpN * WMMA_N;

  // Ensure only the first thread in each warp prints
    // if (threadIdx.x % warpSize == 0 ) {
    //   printf("aRow: %d, aCol: %d, bRow: %d, bCol: %d, i: %d\n", aRow, aCol,
    //          bRow, bCol, i);
    // }
    // printf("aRow: %d, aCol: %d, bRow: %d, bCol: %d, i: %d\n", aRow, aCol,
    // bRow, bCol, i); printf("---end---target---sp--\n");

    // Bounds checking
    if (aRow < M && aCol < K && bRow < K && bCol < N) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);

      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cRow = warpM * WMMA_M;
  int cCol = warpN * WMMA_N;
  // Ensure only the first thread in each warp prints

  if (cRow < M && cCol < N) {
    // if (threadIdx.x % warpSize == 0 ) {
    //     printf("cRow= %d, cCol = %d\n", cRow, cCol);
    // }
   wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

#pragma unroll
    // printf("wmma_example: cRow = %d, cCol = %d\n", cRow, cCol);
    for (int i = 0; i < c_frag.num_elements; i++) {
    // Calculate the exact row and column each fragment element corresponds to
    //   int elemRow = cRow + (i / WMMA_N);
    //   int elemCol = cCol + (i % WMMA_N);
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      // print out current rol and col
    //   if (threadIdx.x % warpSize == 0) {
    //     printf("index: %d\n", i);
    //     printf("Processing element at row %d, col %d, value: %f\n", elemRow, elemCol, c_frag.x[i]);
    //   }
      
      // print current index
    //   if (threadIdx.x % warpSize == 0 ) {
    //     printf("wmma_example: i = %d\n", i);
    //   }
    }

    /// Store the output
    wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);

    // Ensure only the first thread in each warp prints
    if (threadIdx.x % warpSize == 0 ) {
        printf("wmma_example: warpM = %d, warpN = %d\n", warpM, warpN);
    }
  }
}

__global__ void convertFp32ToFp16(half *out, float *in, int n) {
  // count cycles

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n) {
    out[idx] = in[idx];
  }
}

int main(int argc, char *argv[]) {
  // Initialize the problem
  int nkernels = 2;             // number of concurrent kernels
  int nstreams = nkernels + 1;  // use one more stream than concurrent kernel
  // int nbytes = nkernels * sizeof(float_t);  // number of data bytes
  float kernel_time = 10;             // time the kernel should run in ms
  float elapsed_time, elapsed_time2;  // timing variables
  int cuda_device = 0;

  hipDeviceProp_t deviceProp;
  cudaErrCheck(hipGetDevice(&cuda_device));

  cudaErrCheck(hipGetDeviceProperties(&deviceProp, cuda_device));

  if ((deviceProp.concurrentKernels == 0)) {
    printf("> GPU does not support concurrent kernel execution\n");
    printf("  CUDA kernel runs will be serialized\n");
  } else {
    printf("concurrent kernel: %d\n", deviceProp.concurrentKernels);
  }

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    //evaluate the performance
    hipEvent_t start, stop;
    float elapsedTime;
    // cudaEvent_t start1, stop1, start2, stop2;
    // float elapsedTime1, elapsedTime2;   
  //--------------------
  // cuda core
  int N = MATRIX_N;  // Define the size of the matrix
  size_t size = N * N * sizeof(float_t);
  float *h_A, *h_B, *h_C;           // host copies of A, B, C
  float *d_A, *d_B, *d_C;           // device copies of A, B, C

  // float *a_fp32;
  // float *b_fp32;
  half *a_fp16;
  half *b_fp16;
  // //printf("WMMA Example2\n");

  float *c;
  //    float *c_cublas;
  float *c_wmma;

  //    float *c_host_cublas;
  float *c_host_wmma;
  // //printf("WMMA Example3\n");

  // cuda core: Allocate space for host copies and setup values
  cudaErrCheck(hipHostMalloc((void **)&h_A, size, hipHostMallocDefault));
  cudaErrCheck(hipHostMalloc((void **)&h_B, size, hipHostMallocDefault));
  cudaErrCheck(hipHostMalloc((void **)&h_C, size, hipHostMallocDefault));

  // Allocate space for device copies
  cudaErrCheck(hipMalloc((void **)&d_A, size));
  cudaErrCheck(hipMalloc((void **)&d_B, size));
  cudaErrCheck(hipMalloc((void **)&d_C, size));
  cudaErrCheck(
      hipMalloc((void **)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
  cudaErrCheck(
      hipMalloc((void **)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

  // Initialize matrices A and B with random values
  for (int i = 0; i < N * N; i++) {
    h_A[i] = (float)rand() / (float)RAND_MAX *
             10.0;  // Assign a random float value between 0 and 100
    h_B[i] = (float)rand() / (float)RAND_MAX *
             10.0;  // Assign a random float value between 0 and 100
    h_C[i] = 1.5;
  }
  // stream create
  // allocate and initialize an array of stream handles
  hipStream_t *streams =
      (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  hipStreamCreateWithPriority(&streams[0], hipStreamNonBlocking, 0);
  hipStreamCreateWithPriority(&streams[1], hipStreamNonBlocking, 1);
  hipStreamCreateWithPriority(&streams[2], hipStreamNonBlocking, 2);
  // for (int i = 1; i < nstreams; i++) {
  //   cudaErrCheck(cudaStreamCreate(&(streams[i])));
  // }

  // Copy inputs to device
  cudaErrCheck(
      hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, streams[0]));
  cudaErrCheck(
      hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, streams[1]));
  cudaErrCheck(
      hipMemcpyAsync(d_C, h_C, size, hipMemcpyHostToDevice, streams[2]));
  
  //! concurrent

  printf("Converting to fp16...a_fp16\n");
  convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 31) / 32, 32,0,0>>>(
      a_fp16, d_A, MATRIX_M * MATRIX_K);
  //! concurrent
  printf("Converting to fp16...b_fp16\n");
 
  convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 31) / 32, 32,0,0>>>(
      b_fp16, d_B, MATRIX_K * MATRIX_N);
 

  float alpha = 2.0f;
  float beta = 2.0f;
  half alpha_fp16 = 2.0;
  half beta_fp16 = 2.0;

  // printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M,
  // MATRIX_N, MATRIX_K, alpha, beta);

  // First: using WMMA
  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
  //* original
  blockDim.x = 128;
  blockDim.y = 4;
  gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
  gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  //* revised
  // blockDim.x = 32;
  // blockDim.y = 8; 
  // gridDim.x =(MATRIX_M + (WMMA_M * blockDim.x / 64 - 1)) / (WMMA_M * blockDim.x / 64);
  // gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  printf("wmma_example: gridDim.x = %d, gridDim.y = %d\n", gridDim.x,gridDim.y);

// Create events
//! pure tc
hipEventCreate(&start);
hipEventCreate(&stop);
 hipEventRecord(start, 0);

//! tc + sp
// cudaEventCreate(&start1);
// cudaEventCreate(&stop1);
// cudaEventCreate(&start2);
// cudaEventCreate(&stop2);


// Record the start time for the first kernel on stream 0
// cudaEventRecord(start1, streams[0]);
 // Create events for both streams


  // wmma_example<<<gridDim, blockDim, 0, 0>>>(
  //     a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  wmma_example<<<gridDim, blockDim, 0, streams[0]>>>(
      a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

    // cudaEventRecord(stop1, streams[0]);

  dim3 sp_blockDim(16,16);  // Commonly used block size for matrix multiplication
  dim3 sp_gridDim;
  sp_gridDim.x = (N + sp_blockDim.x - 1) / sp_blockDim.x;
  sp_gridDim.y = (N + sp_blockDim.y - 1) / sp_blockDim.y;
  printf("sp_example's : gridDim.x = %d, gridDim.y = %d\n", sp_gridDim.x,
         sp_gridDim.y);
//   sp_example<<<sp_gridDim, sp_blockDim, 0,  streams[0]>>>(
//       a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

// Record the start time for the second kernel on stream 1
    // cudaEventRecord(start2, streams[1]);
    sp_example<<<sp_gridDim, sp_blockDim, 0, streams[1]>>>(
        a_fp16, b_fp16, d_C, MATRIX_M, MATRIX_N, MATRIX_K, alpha_fp16, beta_fp16);
    
    // cudaEventRecord(stop2, streams[1]);
  

  hipStreamSynchronize(streams[0]);
  hipStreamSynchronize(streams[1]);
//   cudaStreamSynchronize(streams[2]);
// Record the end time

//! pure tc
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime, start, stop);
printf("Time for WMMA kernel: %f ms\n", elapsedTime);
hipEventDestroy(start);
hipEventDestroy(stop);
//! tc + sp
// cudaEventSynchronize(stop1);
// cudaEventSynchronize(stop2);
// // Calculate the elapsed time for each kernel
// cudaEventElapsedTime(&elapsedTime1, start1, stop1);
// cudaEventElapsedTime(&elapsedTime2, start2, stop2);
// printf("Time for TC+SP kernel: %f ms\n", elapsedTime1+elapsedTime2);

// Destroy events
// cudaEventDestroy(start1);
// cudaEventDestroy(stop1);
// cudaEventDestroy(start2);
// cudaEventDestroy(stop2);

  hipDeviceSynchronize();

  // Error checking
  printf("\nChecking results...\n");
  //    cudaErrCheck(cudaMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N *
  //    sizeof(float), cudaMemcpyDeviceToHost));

  //* sequential
  //    cudaErrCheck(cudaMemcpy(h_C, d_C, size, cudaMemcpyDeviceToHost));
  //! concurrent
  cudaErrCheck(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));


  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  hipStreamDestroy(streams[2]);
  free(streams);

  cudaErrCheck(hipFree(a_fp16));
  cudaErrCheck(hipFree(b_fp16));


  // cuda core
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  // cudaFree(d_tmpC);
  //* Pinned Memory
  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(h_C);

  exit(EXIT_SUCCESS);
}